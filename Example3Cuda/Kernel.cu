#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "hip/device_functions.h"
__global__ void kernel(unsigned* src, int width, int height, double a, double b) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x < width) && (y < height))
	{
		unsigned pixel = src[y * width + x];
		unsigned red = (pixel & 0x00FF0000) >> 0x10;
		unsigned green = (pixel & 0x0000FF00) >> 0x8;
		unsigned blue = pixel & 0x000000FF;

		unsigned enhancedRed = (a * red + b);
		if (enhancedRed > 255)
			enhancedRed = 255;
		unsigned enhancedGreen = (a * green + b);
		if (enhancedGreen > 255)
			enhancedGreen = 255;
		unsigned enhancedBlue = (a * blue + b);
		if (enhancedBlue > 255)
			enhancedBlue = 255;

		src[y * width + x] = 0xFF000000 | (enhancedRed << 0x10) |
			(enhancedGreen << 0x8) | enhancedBlue;
	}
}

int calc(unsigned* srcImage, int width, int height, double a, double b)
{

	unsigned* dev_srcImage;
	size_t size = sizeof(unsigned) * width * height;
	if (hipMalloc((void**)&dev_srcImage, size) !=
		hipError_t::hipSuccess) return -1;
	if (hipMemcpy(dev_srcImage, srcImage, size,
		hipMemcpyHostToDevice) != hipError_t::hipSuccess)
		return -1;
	// ������������ ���������� ����� �� ���� �����
		dim3 threads(128, 128);
	dim3 blocks((width + threads.x - 1) / threads.x,
		(height + threads.y - 1) / threads.y);
	kernel << <threads, blocks >> > (dev_srcImage, width, height, a, b);
	hipError_t error = hipMemcpy(srcImage, dev_srcImage,
		size,
		hipMemcpyDeviceToHost);
	if (error != hipError_t::hipSuccess) return -1;
	hipFree(dev_srcImage);
	return 0;
}